#include "hip/hip_runtime.h"
#include "kernel.cuh"

#define SEED_RANGE_MAX 10000

__device__ int64_t traverseMap(Mapping **mapsCuds, int *numMaps, int64_t s, int map){

    int nMaps = *(numMaps + map);

    for(int i = 0; i < nMaps; i++){
        Mapping m = *(*(mapsCuds + map) + i);
        if (s >= m.from && s < m.from + m.range) { return m.to + (s - m.from); }
    }

    return s;
}

__global__ void run(int numSeeds, int64_t *seeds, Mapping **mapsCuda, int *numMaps, int64_t *mins){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int64_t min = INT64_MAX;
    for(int i = index; i < numSeeds; i+=stride){
        int64_t startSeed = *(seeds + 2 * i);
        int64_t range = *(seeds + 2 * i + 1);
        for(int64_t j = 0; j < range; j++){
            int64_t outSeed = traverseMap(mapsCuda, numMaps, traverseMap(mapsCuda, numMaps, traverseMap(mapsCuda, numMaps, traverseMap(mapsCuda, numMaps, traverseMap(mapsCuda, numMaps, traverseMap(mapsCuda, numMaps, traverseMap(mapsCuda, numMaps, startSeed + j, 0), 1), 2), 3), 4), 5), 6);
            
            if(outSeed < min){ min = outSeed; }

        }
        *(mins + i) = min;
    }
}

int64_t pt2(std::vector<int64_t>& seedsIn, const std::vector<std::vector<Mapping>>& maps){
    
    std::vector<int64_t> seeds;
    // Split seed ranges up so less than SEED_RANGE_MAX
    for(int i = 0; i < seedsIn.size() / 2; i++){
        while(seedsIn[2 * i + 1] > SEED_RANGE_MAX){
            seeds.push_back(seedsIn[2 * i]);
            seeds.push_back(SEED_RANGE_MAX);
            seedsIn[2 * i] += SEED_RANGE_MAX;
            seedsIn[2 * i + 1] -= SEED_RANGE_MAX;
        }
        seeds.push_back(seedsIn[2 * i]);
        seeds.push_back(seedsIn[2 * i + 1]);
    }
    
    int N = seeds.size() / 2;
    int64_t *seedsCUDA;
    hipMallocManaged(&seedsCUDA, 2 * N * sizeof(int64_t));
    for(int i = 0; i < seeds.size(); i++){
        seedsCUDA[i] = seeds[i];
    }

    Mapping **mapsCUDA;
    hipMallocManaged(&mapsCUDA, maps.size() * sizeof(Mapping *));
    int *numMaps;
    hipMallocManaged(&numMaps, 7 * sizeof(int));
    for(int i = 1; i < 8; i++){
        hipMallocManaged(&mapsCUDA[i - 1], maps[i].size() * sizeof(Mapping));
        numMaps[i - 1] = maps[i].size();
    
        for(int j = 0; j < maps[i].size(); j++){
            mapsCUDA[i - 1][j] = maps[i][j];
        }
    }

    int64_t *mins;
    hipMallocManaged(&mins, N * sizeof(int64_t));

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    run<<<numBlocks, blockSize>>>(N, seedsCUDA, mapsCUDA, numMaps, mins);
    hipDeviceSynchronize();

    int64_t minLoc = INT64_MAX;
    for(int i = 0; i < N; i++){
        if(*(mins + i) < minLoc){ minLoc = *(mins + i); }
    }

    hipFree(seedsCUDA);
    for(int i = 0; i < 7; i++){
        hipFree(mapsCUDA[i]);
    }
    hipFree(mapsCUDA);
    hipFree(numMaps);
    hipFree(mins);

    return minLoc;
}
